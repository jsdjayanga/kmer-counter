#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>
#include "GPUHandler.h"

__global__ void bitEncode(char* input, char* filter, int64_t lineLength,
		int64_t upperBound) {
	uint64_t index = (blockIdx.x * blockDim.x + threadIdx.x) * lineLength;

	if (index > upperBound - lineLength) {
		return;
	}

	uint64_t readValue = 0;
	uint64_t filterValue = 0;

	uint16_t readValueLength = 0;
	uint16_t filterValueLength = 0;

	int64_t i = index;
	for (; i < index + lineLength; i++) {
		if (i - index > 0 && (i - index) % 32 == 0) {
			int64_t readValueLocation = ((((i - index) / 32) - 1)
					* sizeof(int64_t)) + sizeof(int16_t) + index;
			//int64_t filterLocation = ((((i - index) / 32) - 1) * sizeof (int64_t));
			memcpy(&input[readValueLocation], &readValue, sizeof(int64_t));
			//memcpy(&filter[filterLocation], &filterValue, sizeof (int64_t));
			readValue = 0;

			//filterValue = 0;
		}

		if (i - index > 0 && (i - index) % 64 == 0) {
			int64_t filterLocation =
					((((i - index) / 64) - 1) * sizeof(int64_t)) + index;
			memcpy(&filter[filterLocation], &filterValue, sizeof(int64_t));
			filterValue = 0;
		}

		switch (input[i]) {
		case 'A':
			readValue <<= 2;
			readValue |= 0;
			readValueLength += 2;

			filterValue <<= 1;
			filterValueLength += 1;

			continue;
		case 'C':
			readValue <<= 2;
			readValue |= 1;
			readValueLength += 2;

			filterValue <<= 1;
			filterValueLength += 1;

			continue;
		case 'G':
			readValue <<= 2;
			readValue |= 2;
			readValueLength += 2;

			filterValue <<= 1;
			filterValueLength += 1;

			continue;
		case 'T':
			readValue <<= 2;
			readValue |= 3;
			readValueLength += 2;

			filterValue <<= 1;
			filterValueLength += 1;

			continue;
		default:
			readValue <<= 2;
			readValue |= 3;
			readValueLength += 2;

			filterValue <<= 1;
			filterValue |= 1;
			filterValueLength += 1;
			continue;
		}
	}

	//printf("readValueLength==============:%"PRIu16"\n", readValueLength);
	memcpy(&input[index], &readValueLength, sizeof(uint16_t));

	if (i > 0 && (i - index) % 64 > 0) {
		uint8_t shiftingReadValue = (32 - ((i - index) % 32)) * 2;
		readValue <<= shiftingReadValue;

		int64_t readValueLocation = ((((i - index) / 32)) * sizeof(int64_t))
				+ sizeof(int16_t) + index;
		memcpy(&input[readValueLocation], &readValue, sizeof(int64_t));
		readValue = 0;
		readValueLength = 0;

		uint8_t shiftingFilterValue = 64 - ((i - index) % 64);
		filterValue <<= shiftingFilterValue;
		int64_t filterLocation = ((((i - index) / 64)) * sizeof(int64_t))
				+ index;
		memcpy(&filter[filterLocation], &filterValue, sizeof(int64_t));
		filterValue = 0;
		filterValueLength = 0;
	}

}

__device__ bool checkBit(uint64_t filter, uint8_t bit) {
	//cout << "++++++++++:" << (int32_t)bit << endl;
	uint64_t t = 1;
	uint64_t temp = t << (64 - bit - 1);
	if ((temp & filter) > 0) {
		return true;
	}
	return false;
}

__device__ uint64_t read64bits(char* input, int64_t index) {
	uint64_t value = 0;
	memcpy(&value, &input[index], sizeof(int64_t));
	return value;
}

__global__ void extractKMers(char* input, char* bitFilter, char*output,
		uint64_t sectionLength, int64_t kmerLength, int64_t upperBound,
		int64_t lineLength) {
	uint64_t index = (blockIdx.x * blockDim.x + threadIdx.x) * sectionLength;
	uint64_t filterIndex = (blockIdx.x * blockDim.x + threadIdx.x) * lineLength;

	if (filterIndex > upperBound - lineLength) {
		return;
	}

//printf("================================extract=========================================:index=%"PRIu64"\n", filterIndex);

//	uint64_t* f1 = (uint64_t*) &temp[0];
//	uint64_t* f2 = (uint64_t*) &temp[8];
//	uint64_t* f3 = (uint64_t*) &temp[16];
//	cout << *f1 << "|" << *f2 << "|" << *f3 << endl;

	bool validEntry = false;
	int64_t lastInvalidIndex = -1;

	uint16_t i1 = *(uint16_t*) &input[filterIndex];
	uint16_t filterLength = i1 / 2;

	char* encodedInput = &input[filterIndex + sizeof(uint16_t)];

//	cout << "=========:filter length=" << filterLengrh << endl;
	int64_t filterReadLength = 0;
//printf("=======================b4 for loop, filterIndex=%"PRIu64", filterLength=%"PRIu64"\n", filterIndex, filterLength);
	uint64_t outputIndex = 0;
	//uint64_t i = filterIndex;
	for (uint64_t i = 0; i < filterLength; i++) {

//printf("=======================inside for loopindex=%"PRIu64"\n", filterIndex);
		uint64_t filter = 0;
		if (i == 0) {
			memcpy(&filter, &bitFilter[filterIndex], sizeof(uint64_t));
			//filter = (uint64_t*) &bitFilter[filterIndex];
		} else {
			memcpy(&filter, &bitFilter[(((filterIndex + i) / 64) * 8)],
					sizeof(uint64_t));
			//filter = (uint64_t*) &bitFilter[(((filterIndex + i) / 64) * 8)];
		}
//		cout << "=====filter::::" << *filter << "|" << i << "|"
//				<< checkBit(*filter, (uint8_t) i % 64) << endl;

		if (!checkBit(filter, (uint8_t) i % 64)) {
			filterReadLength++;

			if (filterReadLength >= kmerLength) {
//				cout << "valid kmer" << endl;
//printf("================================validkmeri=%"PRIu64", index=%"PRIu64"\n", i, filterIndex);
				uint64_t firstByte = i - kmerLength + 1; // +1 is needed as 'i' starts with index 0
				int64_t shifting = ((firstByte % 32)) * 2;
				int64_t rightShifting = (32 - (kmerLength % 32)) * 2;

				int64_t firstByteToReadEncodedInput = ((firstByte / 32) * 8);
//printf("==========================================================firstByteToReadEncodedInput %"PRIu64", index=%"PRIu64"\n", firstByteToReadEncodedInput, filterIndex);
				uint16_t kmerByteLength = kmerLength / 4;
				uint64_t kmerByteStoreLength = kmerLength / 4;
				if (kmerLength % 4 > 0) {
					kmerByteLength++;
					kmerByteStoreLength += sizeof(uint64_t);
				}

				uint64_t readValue1 = 0;
				uint64_t readValue2 = 0;

				for (int64_t x = firstByteToReadEncodedInput;
						x < firstByteToReadEncodedInput + kmerByteLength; x +=
								sizeof(uint64_t)) {

					//memcpy(&readValue, (char*) &encodedInput[x], sizeof (uint64_t));
					readValue1 = read64bits(encodedInput, x);
//printf("==========================================================actualReadingIndex %"PRIu64", index=%"PRIu64", readValue1Ori=%"PRIu64", shifting=%"PRIu64"\n", x, filterIndex, readValue1, shifting);
					if (shifting > 0) {
						readValue1 <<= shifting;

						if ((x + sizeof(uint64_t)) * 4 < filterLength) { // To avoid reading beyond the limit of bit encoded input
							readValue2 = read64bits(encodedInput,
									x + sizeof(uint64_t));
							readValue2 >>= (64 - shifting);
						}
						readValue1 |= readValue2;
					}

//					cout << "=============================AAAAAA:" << x << "|"
//							<< readValue1 << "|" << index + outputIndex << endl;

					if (x + sizeof(uint64_t)
							>= firstByteToReadEncodedInput + kmerByteLength) {
						readValue1 >>= rightShifting;
						readValue1 <<= rightShifting;
					}

					memcpy(&output[index + outputIndex], &readValue1,
							sizeof(uint64_t));
//printf("==========================================================readValue1 %"PRIu64", actualOutputIndex=%"PRIu64"\n", readValue1, index + outputIndex);
					outputIndex += sizeof(uint64_t);

					readValue1 = 0;
					readValue2 = 0;
				}
				uint32_t count = 1;
				memcpy(&output[index + outputIndex], &count, sizeof(uint32_t));
				outputIndex += sizeof(uint32_t);

				filterReadLength--;
			}

		} else {
			filterReadLength = 0;
		}
	}
}

uint64_t calculateOutputSize(int64_t inputSize, int64_t lineLength,
		int64_t kmerLength) {
	uint64_t records = inputSize / lineLength;
	uint64_t kmerCount = lineLength - kmerLength + 1;
	uint64_t kmerStoreSize = kmerLength / 4;
	if (kmerLength % 4 > 0) {
		kmerStoreSize++;
	}
	kmerStoreSize += 4;
	return kmerCount * kmerStoreSize * records;
}

int64_t processKMers(const char* input, int64_t kmerLength, int64_t inputSize,
		int64_t lineLength) {
	printf("Processing k-mers klen=%"PRIu64", inSize=%"PRIu64","
	" liLen=%"PRIu64"\n", kmerLength, inputSize, lineLength);
	bool debug = true;

	char* d_input;
	char* d_output;
	char* d_filter;

	uint64_t outputSize = calculateOutputSize(inputSize, lineLength,
			kmerLength);
	printf("Outpout size =============%"PRIu64"\n", outputSize);

	hipMalloc((void **) &d_input, inputSize);
	hipMalloc((void **) &d_output, outputSize);
	hipMalloc((void **) &d_filter, inputSize);

	hipMemcpy(d_input, input, inputSize, hipMemcpyHostToDevice);
	hipMemset(d_output, 0, outputSize);
	hipMemset(d_filter, 0, inputSize);

	int32_t threadCount = 256;
	int32_t count = inputSize / lineLength / threadCount;
	if ((inputSize / lineLength) % threadCount > 0) {
		count++;
	}

	for (int32_t ite = 0; ite < count; ite++) {
		bitEncode<<<1, threadCount>>>(&d_input[threadCount * lineLength * ite],
				&d_filter[threadCount * lineLength * ite], lineLength,
				inputSize);
		hipDeviceSynchronize();

		extractKMers<<<1, threadCount>>>(
				&d_input[threadCount * lineLength * ite],
				&d_filter[threadCount * lineLength * ite],
				&d_output[threadCount * outputSize / (inputSize / lineLength)
						* ite], outputSize / (inputSize / lineLength),
				kmerLength, inputSize, lineLength);
		hipDeviceSynchronize();
	}

	printBitEncodedResult(d_input, d_filter, inputSize, lineLength);

	printKmerResult(d_output, outputSize, kmerLength);

	hipDeviceReset();

	return 0;
}

void printBitEncodedResult(char* d_input, char* d_filter, uint64_t inputSize,
		uint64_t lineLength) {
	char* temp = new char[inputSize];
	memset(temp, 0, inputSize);

	char* tempFilter = new char[inputSize];
	memset(tempFilter, 0, inputSize);

	hipMemcpy(temp, d_input, inputSize, hipMemcpyDeviceToHost);
	hipMemcpy(tempFilter, d_filter, inputSize, hipMemcpyDeviceToHost);

	printf(
			"%"PRIu16" : %"PRIu64", %"PRIu64", %"PRIu64", %"PRIu64", %"PRIu64"\n",
			*(uint16_t*) &temp[0], *(uint64_t*) &temp[2],
			*(uint64_t*) &temp[10], *(uint64_t*) &temp[18],
			*(uint64_t*) &temp[26], *(uint64_t*) &temp[36]);

	for (int i = 0; i < inputSize; i += lineLength) {
		uint16_t* count = (uint16_t*) &temp[i];
		printf("===============Count:%"PRIu16"  %i\n", *count, i);
		int iterations = (*count) / 64;
		if ((*count) % 64 > 0) {
			iterations++;
		}
		for (int j = 2; j < iterations * 8; j += 8) {
			printf("%d : %"PRIu64"\n", j, *((uint64_t*) (&temp[i + j])));
		}
		printf("index:%i %"PRIu64", %"PRIu64", %"PRIu64"\n", i,
				*(uint64_t*) &tempFilter[i], *(uint64_t*) &tempFilter[i + 8],
				*(uint64_t*) &tempFilter[i + 16]);
	}

	//		for (int i = 0; i < inputSize; i += lineLength) {
	//			printf("index:%i %"PRIu64", %"PRIu64", %"PRIu64"\n", i,
	//					*(uint64_t*) &tempFilter[i],
	//					*(uint64_t*) &tempFilter[i + 8],
	//					*(uint64_t*) &tempFilter[i + 16]);
	//		}

	//		for (int i = 0; i < inputSize; i++) {
	//			printf("%c", temp[i]);
	//		}

	//		for (int i = 0; i < inputSize; i++) {
	//			printf("%i", (int8_t)tempFilter[i]);
	//		}
	//		printf("\n");

}

void printKmerResult(char* d_output, uint64_t outputSize, uint64_t kmerLength) {
	char* temp = new char[outputSize];
	memset(temp, 0, outputSize);

	hipMemcpy(temp, d_output, outputSize, hipMemcpyDeviceToHost);

	uint64_t kmerByteLength = kmerLength / 32;
	if (kmerLength % 32 > 0) {
		kmerByteLength += 1;
	}
	kmerByteLength *= 8;

	for (int i = 0; i < outputSize; i += kmerByteLength + 4) {
		int j = 0;
		for (; j < kmerByteLength; j += 8) {
			printf("%"PRIu64" ", *(uint64_t*) &temp[i + j]);
		}
		printf("%"PRIu32"\n", *(uint32_t*) &temp[i + j]);
	}

}
